#include "hip/hip_runtime.h"
// Lab3_ProductTwoVectors.cu : Defines the entry point for the console application.
// Author: �ukasz Pawe� Rabiec (259049)

#include "handlers.h"
#include "stdafx.h"
#include <stdlib.h>

#ifndef __HIPCC__
	#define __HIPCC__
#endif

#include <hip/hip_runtime.h>
#include <>

#define SIZE 1024
#define SIZE_OF_INT SIZE*sizeof(int)

void CudaInit()
{
	// Initialize device
	HANDLE_ERROR(hipSetDevice(0));
}

void FillMatrixes(int* firstVector, int* secondVector)
{
	for (int i = 0; i < SIZE; i++)
	{
		firstVector[i] = 1;
		secondVector[i] = 1;
	}
}

//TODO: Przypomniec na pocz�tku zaj��
__global__ void ProductVectorsAtomic(const int* firstVector, const int* secondVector, int* result)
{
	register int sum = 0;

	for (int i = 0; i < SIZE; i++)
	{
		sum += firstVector[i] * secondVector[i];
	}

	*result = sum;
}

__global__ void ProductVectorsWithSumOnSingleThreadV1(const int* firstVector, const int* secondVector, int* result)
{
	__shared__ int cache[SIZE];
	register int localThreadId = threadIdx.x;

	cache[localThreadId] = firstVector[localThreadId] * secondVector[localThreadId];

	__syncthreads();

	// Sum on single thread (poor version)
	if (threadIdx.x == 0)
	{
		result[blockIdx.x] = 0;

		for (int i = 0; i < blockDim.x; i++)
		{
			result[blockIdx.x] += cache[i];
		}
	}
}

__global__ void ProductVectorsWithSumOnSingleThreadV2(const int* firstVector, const int* secondVector, int* result)
{
	__shared__ int product[SIZE];
	//register int prodGlob = blockDim.x * blockIdx.x + threadIdx.x;
	register int localThreadId = threadIdx.x;

	product[localThreadId] = firstVector[localThreadId] * secondVector[localThreadId];

	__syncthreads();

	// Sum on single thread (rich version)
	if (threadIdx.x == 0)
	{
		register int sum = 0;

		for (int i = 0; i < blockDim.x; i++)
		{
			sum += product[i];
		}

		result[blockIdx.x] = sum;
	}
}

int main()
{
	// Initialize
	CudaInit();

	int *firstVector, *secondVector, *multithreadsResult;
	int *devFirstVector, *devSecondVector, *devResult;
	int result;

	// Allocating memory
	firstVector = (int*)malloc(SIZE_OF_INT);
	secondVector = (int*)malloc(SIZE_OF_INT);
	HANDLE_ERROR(hipMalloc((void**)&devFirstVector, SIZE_OF_INT));
	HANDLE_ERROR(hipMalloc((void**)&devSecondVector, SIZE_OF_INT));
	HANDLE_ERROR(hipMalloc((void**)&devResult, SIZE_OF_INT));

	// Operations
	FillMatrixes(firstVector, secondVector);

	HANDLE_ERROR(hipMemcpy(devFirstVector, firstVector, SIZE_OF_INT, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(devSecondVector, secondVector, SIZE_OF_INT, hipMemcpyHostToDevice));

	// Sum on single thread V1
	ProductVectorsWithSumOnSingleThreadV1<<<1, SIZE>>>(devFirstVector, devSecondVector, devResult);
	HANDLE_ERROR(hipMemcpy(&result, devResult, sizeof(int), hipMemcpyDeviceToHost));
	printf("Product vectors with sum on single thread (v1): %d\n", result);

	// Sum on single thread V2
	ProductVectorsWithSumOnSingleThreadV2 << <1, SIZE >> >(devFirstVector, devSecondVector, devResult);
	HANDLE_ERROR(hipMemcpy(&result, devResult, sizeof(int), hipMemcpyDeviceToHost));
	printf("Product vectors with sum on single thread (v2): %d\n", result);

	getchar();

	hipFree(devFirstVector);
	hipFree(devSecondVector);
	hipFree(devResult);
	free(firstVector);
	free(secondVector);

	return 0;
}

