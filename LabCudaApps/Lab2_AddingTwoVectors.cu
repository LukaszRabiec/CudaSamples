#include "hip/hip_runtime.h"
// Lab2_AddingTwoVectors.cu : Defines the entry point for the console application.
// Author: �ukasz Pawe� Rabiec (259049)

#include "stdafx.h"
#include "stdlib.h"
#include "hip/hip_runtime.h"
#include ""
#include "handlers.h"

#define SIZE 32

void FillMatrixes(int* a, int* b)
{
	for (int i = 0; i < SIZE; i++)
	{
		a[i] = -i;
		b[i] = i * i;
	}
}

__global__ void AddVectors(int* a, int* b, int* c)
{
	int tid = blockIdx.x;

	if (tid < SIZE)
	{
		c[tid] = a[tid] + b[tid];
	}

}

int main()
{
	// Initialize device
	HANDLE_ERROR(hipSetDevice(0));

	// Allocating memory on GPU
	int *a, *b, *c, *dev_a, *dev_b, *dev_c;

	size_t bytes = SIZE * sizeof(int);

	a = (int*)malloc(bytes);
	b = (int*)malloc(bytes);
	c = (int*)malloc(bytes);

	HANDLE_ERROR(hipMalloc((void**)&dev_a, bytes));
	HANDLE_ERROR(hipMalloc((void**)&dev_b, bytes));
	HANDLE_ERROR(hipMalloc((void**)&dev_c, bytes));

	FillMatrixes(a, b);

	HANDLE_ERROR(hipMemcpy(dev_a, a, bytes, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_b, b, bytes, hipMemcpyHostToDevice));

	AddVectors<<<SIZE, 1>>>(dev_a, dev_b, dev_c);

	HANDLE_ERROR(hipMemcpy(c, dev_c, bytes, hipMemcpyDeviceToHost));

	for (int i = 0; i < SIZE; i++)
	{
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}

	getchar();

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	free(a);
	free(b);
	free(c);

	return 0;
}

